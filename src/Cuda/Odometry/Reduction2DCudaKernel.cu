#include "hip/hip_runtime.h"
//
// Created by wei on 10/2/18.
//

#include "Reduction2DCuda.cuh"
#include <Cuda/Geometry/ImageCuda.cuh>

namespace three {

template<typename VecType, typename T>
__global__
void ReduceSum2DKernel(ImageCudaServer<VecType> src, T *sum) {
	__shared__ T local_sum[THREAD_2D_UNIT * THREAD_2D_UNIT];

	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int tid = threadIdx.x + threadIdx.y * blockDim.x;

	/** MUST Guarantee this is 0, even if it is not in an image **/
	for (int i = 0; i < TEST_ARRAY_SIZE; ++i) {
		__syncthreads();
		bool flag = (x >= src.width_ || y >= src.height_);
		{
			local_sum[tid] = flag ? 0 : T(src.get(x, y)[0]);
			__syncthreads();

			/** In our case, THREAD_2D_UNIT is fixed **/
			if (tid < 128) local_sum[tid] += local_sum[tid + 128];
			__syncthreads();
			if (tid < 64) local_sum[tid] += local_sum[tid + 64];
			__syncthreads();
			if (tid < 32) WarpReduceSum<T>(local_sum, tid);

			if (tid == 0) atomicAdd(sum, local_sum[0]);
		}
	}
}

template<typename T>
__device__
inline T BlockReduceSum(T sum) {
	/** How many warps do we have? THREAD_2D_UNIT^2 / WAR_SIZE **/
	static __shared__ T warp_sum[THREAD_2D_UNIT * THREAD_2D_UNIT / WARP_SIZE];

	int thread_id = threadIdx.x + threadIdx.y * blockDim.x;
	int lane_id = thread_id % WARP_SIZE;
	int warp_id = thread_id / WARP_SIZE;

	sum = WarpReduceSumShuffle<T>(sum);
	if (lane_id == 0) {
		warp_sum[warp_id] = sum;
	}
	__syncthreads();

	/**
	 * Only fill in the first warp with values indexed by lane
	 * (not that intuitive)
	 **/
	sum = (thread_id < (THREAD_2D_UNIT * THREAD_2D_UNIT / WARP_SIZE)) ?
		  warp_sum[lane_id] : 0;

	if (warp_id == 0) sum = WarpReduceSumShuffle<T>(sum);

	return sum;
}

template<typename VecType, typename T>
__global__
void ReduceSum2DShuffleKernel(ImageCudaServer<VecType> src, T *sum_total) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	for (int i = 0; i < TEST_ARRAY_SIZE; ++i) {
		T sum =
			(x >= src.width_ || y >= src.height_) ? 0 : T(src.get(x, y)[0]);
		__syncthreads();
		sum = BlockReduceSum(sum);
		if (threadIdx.x == 0) atomicAdd(sum_total, sum);
	}
}


/** Why is it so fast ??? **/
template<typename VecType, typename T>
__global__
void AtomicSumKernel(ImageCudaServer<VecType> src, T *sum_total) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	for (int i = 0; i < TEST_ARRAY_SIZE; ++i) {
		T sum =
			(x >= src.width_ || y >= src.height_) ? 0 : T(src.get(x, y)[0]);
		__syncthreads();
		atomicAdd(sum_total, sum);
	}
}


template
float ReduceSum2D<Vector1f, float>(ImageCuda<Vector1f> &src);

template
float ReduceSum2DShuffle<Vector1f, float>(ImageCuda<Vector1f> &src);

template
float AtomicSum<Vector1f, float>(ImageCuda<Vector1f> &src);

template
int ReduceSum2D<Vector1b, int>(ImageCuda<Vector1b> &src);

template
int ReduceSum2DShuffle<Vector1b, int>(ImageCuda<Vector1b> &src);

template
int AtomicSum<Vector1b, int>(ImageCuda<Vector1b> &src);
}