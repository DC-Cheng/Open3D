/**
 * Created by wei on 18-9-27
 */

#include "UtilsCuda.h"
#include "HelperCuda.h"

namespace three {

void CudaMalloc(void** ptr, size_t size) {
	checkCudaErrors(hipMalloc(ptr, size));
}

void CudaFree(void* ptr) {
	checkCudaErrors(hipFree(ptr));
}

void CudaMemcpy(void *dst, const void *src, size_t size,
	enum MemcpyKind kind) {
	hipMemcpyKind kind_;
	switch (kind) {
		case HostToHost:
			kind_ = hipMemcpyHostToHost; break;
		case HostToDevice:
			kind_ = hipMemcpyHostToDevice; break;
		case DeviceToDevice:
			kind_ = hipMemcpyDeviceToDevice; break;
		case DeviceToHost:
			kind_ = hipMemcpyDeviceToHost; break;
		default:
			/* Should never reach here */
			break;
	}
	checkCudaErrors(hipMemcpy(dst, src, size, kind_));
}

void CudaMemset(void *ptr, int value, size_t size) {
	checkCudaErrors(hipMemset(ptr, value, size));
}

void CudaSynchronize() {
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
}

}