#include "hip/hip_runtime.h"
#include "test_func_lib.h"

// inline void __OPEN3D_CUDA_CHECK(hipError_t err,
//                                 const char *file,
//                                 const int line) {
//     if (err != hipSuccess) {
//         printf("%s:%d CUDA runtime error: %s", file, line,
//                hipGetErrorString(err));
//     }
// }

/// In library to be compiled alone
// typedef int (*hash_t)(int);

// __global__ void Kernel(hash_t func) {
//     int tid = threadIdx.x;
//     int out = (*func)(tid);
//     printf("tid %d -> out %d\n", tid, out);
// }

// class Caller {
// public:
//     void Launch(hash_t h_func) {
//         Kernel<<<1, 10>>>(h_func);
//         __OPEN3D_CUDA_CHECK(hipDeviceSynchronize(), __FILE__, __LINE__);
//         __OPEN3D_CUDA_CHECK(hipGetLastError(), __FILE__, __LINE__);
//     }
// };

/// In source
/// We still need to write some cuda code in main file, but the library doesn't
/// have to see the device function
__device__ int inc(int val) { return val + 1; }
/// This assignment cannot be ignored, otherwise there will be 'invalid program
/// counter' error
__device__ hash_t hash = inc;

int main() {
    hash_t h_func;
    hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(hash), sizeof(hash_t));
    __OPEN3D_CUDA_CHECK(hipGetLastError(), __FILE__, __LINE__);

    Caller caller;
    caller.Launch(h_func);
}